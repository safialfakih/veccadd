#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

__host__ __device__ float max(float a, float b) {
    if (a>b){
        return a;
    }else{
        return b;
    }
}

__global__ void vecMax_kernel(double* a, double* b, double* c, unsigned int M) {
    __global__ void vecadd_kernel(float* x, float* y, float* z, int N) {
        int i = blockDim.x*blockIdx.x + threadIdx.x;
        if (i < N) {
                z[i] = max(x[i], y[i]);
                }
        }
}

void vecMax_gpu(double* a, double* b, double* c, unsigned int M) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);
    float *x_d, *y_d, *z_d;
    hipMalloc((void**) &x_d, N*sizeof(float));
    hipMalloc((void**) &y_d, N*sizeof(float));
    hipMalloc((void**) &z_d, N*sizeof(float));

    // TODO
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);
    hipMemcpy(x_d, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, N*sizeof(float), hipMemcpyHostToDevice);

    // TODO



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);
    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = N/numThreadsPerBlock;
    vecadd_kernel <<< numBlocks, numThreadsPerBlock >>> (x_d, y_d, z_d, N);

    // TODO




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);
    hipMemcpy(z, z_d, N*sizeof(float), hipMemcpyDeviceToHost);

    // TODO


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    // TODO



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

